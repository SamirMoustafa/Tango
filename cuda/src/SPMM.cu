#include "utils.h"
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hiprtc.h>
#include <stdlib.h>
#include <torch/extension.h>

#define CHECK_CUSPARSE(func)                                                   \
    {                                                                          \
        hipsparseStatus_t status = (func);                                      \
        if (status != HIPSPARSE_STATUS_SUCCESS) {                               \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n",     \
                   __LINE__, hipsparseGetErrorString(status), status);          \
            exit(EXIT_FAILURE);                                                \
        }                                                                      \
    }

const auto transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
const auto transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
const auto ALG = HIPSPARSE_SPMM_CSR_ALG2;

void cusp_SPMM_csr_impl(torch::Tensor &out, const torch::Tensor &indptr,
                        const torch::Tensor &indices, const torch::Tensor &N,
                        const torch::Tensor &scale) {
    auto handle = at::cuda::getCurrentCUDASparseHandle();
    hipsparsePointerMode_t ptr_mode;
    hipsparseGetPointerMode(handle, &ptr_mode);
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
    int64_t m = indptr.size(0) - 1;
    int64_t n = out.size(-1);
    int64_t k = N.size(0);
    int64_t nnz = indices.size(0);
    float *alpha = scale.data_ptr<float>();
    auto beta_tensor = torch::zeros_like(scale);
    float *beta = beta_tensor.data_ptr<float>();
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    // sparse matrix A is all one
    auto A = torch::empty({nnz}, torch::dtype(torch::kInt8).device(N.device()));
    int8_t *A_ptr = A.data_ptr<int8_t>();
    cudaErrchk(hipMemset(A_ptr, 1, nnz * sizeof(int8_t)));

    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, m, k, nnz, indptr.data_ptr<int64_t>(),
        indices.data_ptr<int64_t>(), A_ptr, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_8I));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, k, n, n, N.data_ptr<int8_t>(),
                                       HIP_R_8I, HIPSPARSE_ORDER_ROW));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, m, n, n, out.data_ptr<float>(),
                                       HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // workspace
    size_t workspace_size;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, transA, transB, alpha, matA,
                                           matB, beta, matC, HIP_R_32F, ALG,
                                           &workspace_size));
    auto workspace = torch::empty(
        {(long)workspace_size}, torch::dtype(torch::kInt8).device(N.device()));
    int8_t *workspace_ptr = workspace.data_ptr<int8_t>();
    // call SPMM
    CHECK_CUSPARSE(hipsparseSpMM(handle, transA, transB, alpha, matA, matB, beta,
                                matC, HIP_R_32F, ALG, workspace_ptr));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    hipsparseSetPointerMode(handle, ptr_mode);
}

torch::Tensor multi_cusparse_SPMM_broadcastE_impl(const torch::Tensor &indptr,
                                                  const torch::Tensor &indices,
                                                  const torch::Tensor &N,
                                                  const torch::Tensor &E_T,
                                                  const torch::Tensor &scale) {
    const auto &device = N.device();
    float *alpha = scale.data_ptr<float>();
    auto beta_tensor = torch::zeros_like(scale);
    float *beta = beta_tensor.data_ptr<float>();
    auto handle = at::cuda::getCurrentCUDASparseHandle();
    hipsparsePointerMode_t ptr_mode;
    hipsparseGetPointerMode(handle, &ptr_mode);
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
    const int64_t m = indptr.size(0) - 1;
    const int64_t n = N.size(-1);
    const int64_t k = N.size(0);
    const int64_t nnz = indices.size(0);
    const int64_t b = N.size(1);
    // Sparse matrix
    hipsparseSpMatDescr_t matA;
    // E is transposed to (b, nnz, 1) making contiguous memory are edges
    auto A_ptr = E_T.data_ptr<int8_t>();
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, m, k, nnz, indptr.data_ptr<int64_t>(),
        indices.data_ptr<int64_t>(), A_ptr, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_8I));
    // Dense matrix
    hipsparseDnMatDescr_t matB, matC;
    // Output
    auto out =
        torch::empty({m, b, n}, torch::dtype(torch::kFloat32).device(device));
    auto out_ptr = out.data_ptr<float>();
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, m, n, b * n, out_ptr, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));
    // Node features
    auto N_ptr = N.data_ptr<int8_t>();
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, k, n, b * n, N_ptr, HIP_R_8I,
                                       HIPSPARSE_ORDER_ROW));
    // workspace
    size_t workspace_size;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, transA, transB, alpha, matA,
                                           matB, beta, matC, HIP_R_32F, ALG,
                                           &workspace_size));
    auto workspace = torch::empty({(long)workspace_size},
                                  torch::dtype(torch::kInt8).device(device));
    auto *ws_ptr = workspace.data_ptr<int8_t>();
    for (int i = 1; i <= b; i++) {
        // call SPMM
        CHECK_CUSPARSE(hipsparseSpMM(handle, transA, transB, alpha, matA, matB,
                                    beta, matC, HIP_R_32F, ALG, ws_ptr));
        // move to next block
        CHECK_CUSPARSE(hipsparseSpMatSetValues(matA, A_ptr + i * nnz));
        CHECK_CUSPARSE(hipsparseDnMatSetValues(matB, N_ptr + i * n));
        CHECK_CUSPARSE(hipsparseDnMatSetValues(matC, out_ptr + i * n));
    }
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    hipsparseSetPointerMode(handle, ptr_mode);
    return out;
}

torch::Tensor multi_cusparse_SPMM_elementwise_impl(const torch::Tensor &indptr,
                                                   const torch::Tensor &indices,
                                                   const torch::Tensor &N,
                                                   const torch::Tensor &E_T,
                                                   const torch::Tensor &scale) {
    const auto &device = N.device();
    auto handle = at::cuda::getCurrentCUDASparseHandle();
    hipsparsePointerMode_t ptr_mode;
    hipsparseGetPointerMode(handle, &ptr_mode);
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
    float *alpha = scale.data_ptr<float>();
    auto beta_tensor = torch::zeros_like(scale);
    float *beta = beta_tensor.data_ptr<float>();

    const int64_t m = indptr.size(0) - 1;
    const int64_t n = 1;
    const int64_t k = N.size(0);
    const int64_t nnz = indices.size(0);
    const int64_t b = N.size(1) * N.size(2);
    // Sparse matrix
    hipsparseSpMatDescr_t matA;
    // E is transposed making contiguous memory are edges
    auto A_ptr = E_T.data_ptr<int8_t>();
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, m, k, nnz, indptr.data_ptr<int64_t>(),
        indices.data_ptr<int64_t>(), A_ptr, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_8I));
    // Dense matrix
    hipsparseDnMatDescr_t matB, matC;
    // Output
    auto out = torch::empty({m, N.size(1), N.size(2)},
                            torch::dtype(torch::kFloat32).device(device));
    auto out_ptr = out.data_ptr<float>();
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, m, n, b * n, out_ptr, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));
    // Node features
    auto N_ptr = N.data_ptr<int8_t>();
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, k, n, b * n, N_ptr, HIP_R_8I,
                                       HIPSPARSE_ORDER_ROW));
    // workspace
    size_t workspace_size;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, transA, transB, alpha, matA,
                                           matB, beta, matC, HIP_R_32F, ALG,
                                           &workspace_size));
    auto workspace = torch::empty({(long)workspace_size},
                                  torch::dtype(torch::kInt8).device(device));
    auto *ws_ptr = workspace.data_ptr<int8_t>();
    for (int i = 1; i <= b; i++) {
        // call SPMM
        CHECK_CUSPARSE(hipsparseSpMM(handle, transA, transB, alpha, matA, matB,
                                    beta, matC, HIP_R_32F, ALG, ws_ptr));
        // move to next block
        CHECK_CUSPARSE(hipsparseSpMatSetValues(matA, A_ptr + i * nnz));
        CHECK_CUSPARSE(hipsparseDnMatSetValues(matB, N_ptr + i * n));
        CHECK_CUSPARSE(hipsparseDnMatSetValues(matC, out_ptr + i * n));
        kernelErrchk();
    }
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    hipsparseSetPointerMode(handle, ptr_mode);
    return out;
}

torch::Tensor multi_cusparse_SPMV_impl(const torch::Tensor &indptr,
                                       const torch::Tensor &indices,
                                       const torch::Tensor &N_T,
                                       const torch::Tensor &E_T,
                                       const torch::Tensor &scale) {
    const auto &device = N_T.device();
    auto handle = at::cuda::getCurrentCUDASparseHandle();
    hipsparsePointerMode_t ptr_mode;
    hipsparseGetPointerMode(handle, &ptr_mode);
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
    float *alpha = scale.data_ptr<float>();
    auto beta_tensor = torch::zeros_like(scale);
    float *beta = beta_tensor.data_ptr<float>();

    const int64_t m = indptr.size(0) - 1;
    const int64_t n = 1;
    const int64_t k = N_T.size(-1);
    const int64_t nnz = indices.size(0);
    const int64_t b = N_T.size(1) * N_T.size(0);
    // Sparse matrix
    hipsparseSpMatDescr_t matA;
    // E is transposed making contiguous memory are edges
    auto A_ptr = E_T.data_ptr<int8_t>();
    CHECK_CUSPARSE(hipsparseCreateCsr(
        &matA, m, k, nnz, indptr.data_ptr<int64_t>(),
        indices.data_ptr<int64_t>(), A_ptr, HIPSPARSE_INDEX_64I,
        HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_8I));
    // Dense vector
    hipsparseDnVecDescr_t vecB, vecC;
    // Output
    auto out =
        torch::empty({m, b}, torch::dtype(torch::kFloat32).device(device));
    // auto out_ptr = out.data_ptr<float>();
    // CHECK_CUSPARSE(hipsparseCreateDnVec(&vecC, m, out_ptr, HIP_R_32F));
    // // Node features
    // auto N_ptr = N_T.data_ptr<int8_t>();
    // CHECK_CUSPARSE(hipsparseCreateDnVec(&vecB, k, N_ptr, HIP_R_8I));
    // // workspace
    // size_t workspace_size;
    // CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
    //     handle, transA, alpha, matA, vecB, beta, vecC, HIP_R_32F,
    //     HIPSPARSE_SPMV_ALG_DEFAULT, &workspace_size));
    // auto workspace = torch::empty({(long)workspace_size},
    //                               torch::dtype(torch::kInt8).device(device));
    // auto *ws_ptr = (size_t*)workspace.data_ptr<int8_t>();
    // for (int i = 1; i <= b; i++) {
    //     // call SPMM
    //     CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
    //         handle, transA, alpha, matA, vecB, beta, vecC, HIP_R_32F,
    //         HIPSPARSE_SPMV_ALG_DEFAULT, ws_ptr));
    //     // move to next block
    //     CHECK_CUSPARSE(hipsparseSpMatSetValues(matA, A_ptr + i * nnz));
    //     CHECK_CUSPARSE(hipsparseDnVecSetValues(vecB, N_ptr + i * k));
    //     CHECK_CUSPARSE(hipsparseDnVecSetValues(vecC, out_ptr + i * k));
    //     kernelErrchk();
    // }
    // CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    // CHECK_CUSPARSE(hipsparseDestroyDnVec(vecB));
    // CHECK_CUSPARSE(hipsparseDestroyDnVec(vecC));
    // hipsparseSetPointerMode(handle, ptr_mode);
    return out;
}

void incidence_SPMM_impl(torch::Tensor &out, const torch::Tensor &src,
                         const torch::Tensor &dst, const torch::Tensor &E,
                         const torch::Tensor &scale) {
    auto handle = at::cuda::getCurrentCUDASparseHandle();
    hipsparsePointerMode_t ptr_mode;
    hipsparseGetPointerMode(handle, &ptr_mode);

    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
    float *alpha = scale.data_ptr<float>();
    auto beta_tensor = torch::zeros_like(scale);
    float *beta = beta_tensor.data_ptr<float>();

    const int64_t m = out.size(0);
    const int64_t n = out.size(1);
    const int64_t k = E.size(0);
    const int64_t nnz = src.size(0);
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;

    // sparse matrix A is all one
    auto A = torch::empty({nnz}, torch::dtype(torch::kInt8).device(E.device()));
    int8_t *A_ptr = A.data_ptr<int8_t>();
    cudaErrchk(hipMemset(A_ptr, 1, nnz * sizeof(int8_t)));
    CHECK_CUSPARSE(hipsparseCreateCoo(
        &matA, m, nnz, nnz, src.data_ptr<int64_t>(), dst.data_ptr<int64_t>(),
        A_ptr, HIPSPARSE_INDEX_64I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_8I));
    // Dense matrix
    auto E_ptr = E.data_ptr<int8_t>();
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, nnz, n, n, E_ptr, HIP_R_8I,
                                       HIPSPARSE_ORDER_ROW));
    // Output
    auto out_ptr = out.data_ptr<float>();
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, m, n, n, out_ptr, HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));
    // call SPMM
    // std::cout << alpha << " " << A_ptr << " " << src.data_ptr<int64_t>() << "
    // " << dst.data_ptr<int64_t>() << " " << out_ptr << "\n";
    CHECK_CUSPARSE(hipsparseSpMM(handle, transA, transB, alpha, matA, matB, beta,
                                matC, HIP_R_32F, HIPSPARSE_SPMM_COO_ALG4,
                                nullptr));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    hipsparseSetPointerMode(handle, ptr_mode);
}
